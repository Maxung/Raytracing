#include "hip/hip_runtime.h"
#include "Sphere.h"
#include "HittableList.h"
#include "Vec3.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include <cfloat>
#include <ctime>
#include <iostream>
#include "Camera.h"
#include "Material.h"
#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        hipDeviceReset();
        exit(99);
    }
}

__device__ Vec3 color(const Ray& r, Hittable **world, int depth, hiprandState *lrs) {
    Ray cur_ray = r;
    Vec3 cur_attenuation = Vec3(1, 1, 1);
    for (int i = 0; i < 50; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            Ray scattered;
            Vec3 attenuation;
            if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, lrs)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            } else {
                return Vec3(0, 0, 0);
            }
        }
        else { // background
            Vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f * (unit_direction.y() + 1.0f); // scaling to 0.0 <-> 1.0
            Vec3 c = (1.0f - t) * Vec3(1.0, 1.0, 1.0) + t * Vec3(0.5, 0.7, 1.0); // blend
            return cur_attenuation * c;
        }
    }
    return Vec3(0, 0, 0);
}

__global__ void random_scene(Hittable **list,  Hittable **world, Camera **camera, int nx, int ny, hiprandState
        *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState lrs = *rand_state;
        list[0] =  new Sphere(Vec3(0,-1000,0), 1000, new Lambertian(Vec3(0.5, 0.5, 0.5)));
        int i = 1;
        for (int a = -11; a < 11; a++) {
            for (int b = -11; b < 11; b++) {
                float choose_mat = hiprand_uniform(&lrs);
                Vec3 center(a+0.9*hiprand_uniform(&lrs),0.2,b+0.9*hiprand_uniform(&lrs));
                if ((center-Vec3(4,0.2,0)).length() > 0.9) {
                    if (choose_mat < 0.8) {  // diffuse
                        list[i++] = new Sphere(center, 0.2,
                            new Lambertian(Vec3(hiprand_uniform(&lrs)*hiprand_uniform(&lrs),
                                            hiprand_uniform(&lrs)*hiprand_uniform(&lrs),
                                            hiprand_uniform(&lrs)*hiprand_uniform(&lrs))
                            )
                        );
                    }
                    else if (choose_mat < 0.95) { // Metal
                        list[i++] = new Sphere(center, 0.2,
                                new Metal(Vec3(0.5*(1 + hiprand_uniform(&lrs)),
                                           0.5*(1 + hiprand_uniform(&lrs)),
                                           0.5*(1 + hiprand_uniform(&lrs))),
                                           0.5*hiprand_uniform(&lrs)));
                    }
                }
            }
        }

        list[i++] = new Sphere(Vec3(4, 1, 0), 1.0, new Metal(Vec3(0.7, 0.6, 0.5), 0.0));
        list[i++] = new Sphere(Vec3(-4, 1, 0), 1.0, new Lambertian(Vec3(0.4, 0.2, 0.1)));

        *world = new HittableList(list,i);
        *camera = new Camera(Vec3(13, 2, 3), Vec3(0, 0, 0), Vec3(0, 1, 0), 20, float(nx) / float(ny));
    }
}

__global__ void rand_init(hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0)
        hiprand_init(1984, 0, 0, rand_state);
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(Vec3 *fb, int max_x, int max_y,  hiprandState *rand_state, Camera **cam, int ns,
        Hittable **world) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    hiprandState lrs = rand_state[pixel_index];
    Vec3 col(0, 0, 0);
    for (int s = 0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&lrs)) / float(max_x);
        float v = float(j + hiprand_uniform(&lrs)) / float(max_y);
        Ray r = (*cam)->get_ray(u, v);
        col += color(r, world, 0, &lrs);
    }
    col /= float(ns);
    col = Vec3(sqrt(col[0]), sqrt(col[1]), sqrt(col[2]));

    fb[pixel_index] = col;
}

__global__ void cleanup(Hittable **list, Hittable **world,  Camera **camera) {
    for(int i = 0; i < 500; i++) {
        delete (reinterpret_cast<Sphere*>(list[i])->mat_ptr);
        delete list[i]; 
    }
    delete *world;
    delete *camera;
}

int main() {
    const int nx = 2000;
    const int ny = 1000;
    int ns = 10;
    unsigned char image[nx * ny * 3]; // RGB image
    Vec3 *fb;
    Hittable **list;
    Hittable **world;
    Camera **camera;
    std::clock_t start = std::clock();
	
    int tx = 8;
    int ty = 8;
    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);
    hiprandState *rand_state;
    hiprandState *rand_state2;

    checkCudaErrors(hipMallocManaged(reinterpret_cast<void**>(&fb), nx * ny * sizeof(Vec3)));
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&rand_state), nx * ny * sizeof(hiprandState)));
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&rand_state2), 1 * sizeof(hiprandState)));
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&list), 500 * sizeof(Hittable *)));
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&world), sizeof(Hittable *)));
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&camera), sizeof(Camera *)));

    rand_init<<<1, 1>>>(rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    random_scene<<<1, 1>>>(list, world, camera, nx, ny, rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    render_init<<<blocks, threads>>>(nx, ny, rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    render<<<blocks, threads>>>(fb, nx, ny, rand_state, camera, ns, world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    int index = 0;
    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j * nx + i;
            int ir = int(255.99f * fb[pixel_index].r());
            int ig = int(255.99f * fb[pixel_index].g());
            int ib = int(255.99f * fb[pixel_index].b());
            image[index++] = ir;
            image[index++] = ig;
            image[index++] = ib;
        }
    }
	
    std::clock_t end = std::clock();
    std::cout << "CPU time: " << 1000.0 * (end - start) / CLOCKS_PER_SEC << "ms" << std::endl;    

    stbi_write_jpg("render.jpg", nx, ny, 3, image, 100);

    checkCudaErrors(hipDeviceSynchronize());
    cleanup<<<1, 1>>>(list, world, camera);
    checkCudaErrors(hipGetLastError());
    //checkCudaErrors(hipFree(camera));
    //checkCudaErrors(hipFree(world));
    //checkCudaErrors(hipFree(list));
    //checkCudaErrors(hipFree(rand_state));
    
    hipDeviceReset();
}
